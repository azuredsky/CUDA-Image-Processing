#include "cudaImageDevice.h.cu"


// Initialize a master list that
list<cudaImageDevice*> cudaImageDevice::masterDevImageList_ = list<cudaImageDevice*>(0);
int cudaImageDevice::totalDevMemUsed_ = 0;


////////////////////////////////////////////////////////////////////////////////
void cudaImageDevice::Allocate(int ncols, int nrows)
{
   imgCols_ = ncols;
   imgRows_ = nrows;
   imgElts_ = imgCols_*imgRows_;
   imgBytes_ = imgElts_*sizeof(int);

   if(ncols == 0 || nrows == 0)
      imgData_ = NULL;
   else
   {
      hipMalloc((void**)&imgData_, imgBytes_);
      totalDevMemUsed_ += imgBytes_;

      masterDevImageList_.push_back(this);
      trackingIter_ = masterDevImageList_.end();  // this is one-past-the-end
      trackingIter_--;

      static int idVal = 99;
      idVal++;
      id_ = idVal;
   }
}

////////////////////////////////////////////////////////////////////////////////
void cudaImageDevice::Deallocate(void)
{
   if(imgData_ != NULL)
   {
      hipFree(imgData_);
      totalDevMemUsed_ -= imgBytes_;
      if(trackingIter_ != masterDevImageList_.end())
      {
         masterDevImageList_.erase(trackingIter_);
         trackingIter_ = masterDevImageList_.end();
      }
   } 
   imgData_ = NULL;
   imgCols_ = imgRows_ = imgElts_ = imgBytes_ = 0;
}

////////////////////////////////////////////////////////////////////////////////
void cudaImageDevice::resize(int ncols, int nrows)
{
   if( imgElts_ == ncols*nrows)
   {
      // imgElts_ and imgBytes_ is already correct, no need to realloc
      imgCols_ = ncols;
      imgRows_ = nrows;
   }
   else
   {
      Deallocate();
      Allocate(ncols, nrows);
   }
}

////////////////////////////////////////////////////////////////////////////////
cudaImageDevice::~cudaImageDevice()
{
   Deallocate();
}

////////////////////////////////////////////////////////////////////////////////
cudaImageDevice::cudaImageDevice() :
   imgData_(NULL), imgCols_(0), imgRows_(0), imgElts_(0), imgBytes_(0) { }

////////////////////////////////////////////////////////////////////////////////
cudaImageDevice::cudaImageDevice(int ncols, int nrows) :
   imgData_(NULL), imgCols_(0), imgRows_(0), imgElts_(0), imgBytes_(0)
{
   Allocate(ncols, nrows);
}


////////////////////////////////////////////////////////////////////////////////
cudaImageDevice::cudaImageDevice(cudaImageHost const & hostImg) :
   imgData_(NULL), imgCols_(0), imgRows_(0), imgElts_(0), imgBytes_(0)
{
   copyFromHost(hostImg);
}

////////////////////////////////////////////////////////////////////////////////
cudaImageDevice::cudaImageDevice(cudaImageDevice const & devImg) :
   imgData_(NULL), imgCols_(0), imgRows_(0), imgElts_(0), imgBytes_(0)
{
   copyFromDevice(devImg);
}




////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// 
// MEMORY COPY WRAPPERS
//
// These 8 methods handle all the possible ways we might want to copy data in,
// out, or between device memory locations.  
//
// NOTE:  These methods are not designed to explicitly allocate anyone else's
//        memory, so if we have only a pointer to destination memory, we have 
//        to assume it is already allocated properly.
//
//        If we're passed a reference to a cudaImage, we will call resize()
//        before copying to it.
//
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// HOST <---> DEVICE 
/////
void cudaImageDevice::copyFromHost  (int* hostPtr, int ncols, int nrows)
{
   resize(ncols, nrows);
   hipMemcpy(imgData_, hostPtr, imgBytes_, hipMemcpyHostToDevice);
}

/////
void cudaImageDevice::copyFromHost  (cudaImageHost const & hostImg)
{
   copyFromHost(hostImg.getDataPtr(), hostImg.numCols(), hostImg.numRows());
}

/////
void cudaImageDevice::copyToHost(int* hostPtr) const
{
   hipMemcpy(hostPtr, imgData_, imgBytes_, hipMemcpyDeviceToHost);
}

/////
void cudaImageDevice::copyToHost(cudaImageHost & hostImg) const
{
   hostImg.resize(imgCols_, imgRows_);
   copyToHost(hostImg.getDataPtr());
}

////////////////////////////////////////////////////////////////////////////////
// DEVICE <---> DEVICE 
/////
void cudaImageDevice::copyFromDevice(int* devicePtr, int ncols, int nrows)
{
   resize(ncols, nrows);
   hipMemcpy(imgData_, devicePtr, imgBytes_, hipMemcpyDeviceToDevice);
}

/////
void cudaImageDevice::copyFromDevice(cudaImageDevice const & devImg)
{
   copyFromDevice(devImg.getDataPtr(), devImg.numCols(), devImg.numRows());
}

/////
void cudaImageDevice::copyToDevice(int* devPtr) const
{
   hipMemcpy(devPtr, imgData_, imgBytes_, hipMemcpyDeviceToDevice);
}

/////
void cudaImageDevice::copyToDevice(cudaImageDevice & devImg) const
{
   devImg.resize(imgCols_, imgRows_);
   copyToDevice(devImg.getDataPtr());
}




////////////////////////////////////////////////////////////////////////////////
int cudaImageDevice::calculateDeviceMemoryUsage(bool dispStdout)
{
   float sizeMB = 1024.0f * 1024.0f;
   int totalBytes = 0;
   int nimg = (int)masterDevImageList_.size();
   int ct = 0;


   if(dispStdout)
      printf("\tDevice memory contains _%d_ cudaImageDevice objects\n", nimg);

   list<cudaImageDevice*>::iterator it;
   for( it  = masterDevImageList_.begin();
        it != masterDevImageList_.end();
        it++)
   {
      int nbytes = (*it)->imgBytes_;
      int wholeMB = nbytes / sizeMB;
      int fracMB  = (int)(10000 * (float)(nbytes - wholeMB*sizeMB) / (float)sizeMB);
      if(dispStdout)
         printf("\t\tDevice Image %3d (ID=%03d):  %4d x %4d,   %4d.%04d MB\n", ct, (*it)->id_, (*it)->imgCols_, (*it)->imgRows_, wholeMB, fracMB);
                                   
      ct++;

      totalBytes += (*it)->imgBytes_;
   }

   if(dispStdout)
   {
      int wholeMB = totalBytes / sizeMB;
      int fracMB  = (int)(10000 * (float)(totalBytes - wholeMB*sizeMB) / (float)sizeMB);
      printf("\t\t-------------------------------------------------------\n");
      printf("\t\tTotal Device Memory Used:                  %4d.%04d MB\n\n", wholeMB, fracMB);
   }
   
   return totalDevMemUsed_;
}

void cudaImageDevice::printMask(char zero, char one)
{
   cudaImageHost hptr;
   copyToHost(hptr);
   hptr.printMask(zero, one);
}




